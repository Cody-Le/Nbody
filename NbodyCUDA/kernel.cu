#include "hip/hip_runtime.h"
﻿


//Modifed version of simple CUDA nbody simulation from github



#include "hip/hip_runtime.h"
#include ""
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <string>
#include "timer.h"

#define BLOCK_SIZE 256
#define SOFTENING 1e-9f

typedef struct { float x, y, vx, vy; } Body;


void randomizeBodies(float* data, int n) {
    for (int i = 0; i < n; i++) {
        data[i] = 2.0f * (rand() / (float)RAND_MAX) - 1.0f;
    }
}

__global__
void bodyForce(Body* p, float dt, int n) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < n) {
        float Fx = 0.0f; float Fy = 0.0f;

        for (int j = 0; j < n; j++) {
            float dx = p[j].x - p[i].x;
            float dy = p[j].y - p[i].y;
          
            float distSqr = dx * dx + dy * dy + SOFTENING;
            float invDist = sqrt(distSqr);
            float invDist3 = invDist * invDist * invDist;

            Fx += dx * invDist3; Fy += dy * invDist3; 
        }

        p[i].vx += dt * Fx; p[i].vy += dt * Fy;
    }
}

int main(const int argc, const char** argv) {

    int nBodies = 1000;
    if (argc > 1) nBodies = atoi(argv[1]);

    const float dt = 0.01f; // time step
    const int nIters = 2000;  // simulation iterations

    int bytes = nBodies * sizeof(Body);
    float* buf = (float*)malloc(bytes);
    Body* p = (Body*)buf;

    randomizeBodies(buf, 4 * nBodies); // Init pos / vel data

    float* d_buf;
    hipMalloc(&d_buf, bytes);
    Body* d_p = (Body*)d_buf;

    int nBlocks = (nBodies + BLOCK_SIZE - 1) / BLOCK_SIZE;
    double totalTime = 0.0;

    for (int iter = 1; iter <= nIters; iter++) {
        StartTimer();

        hipMemcpy(d_buf, buf, bytes, hipMemcpyHostToDevice);
        bodyForce << <nBlocks, BLOCK_SIZE >> > (d_p, dt, nBodies); // compute interbody forces
        hipMemcpy(buf, d_buf, bytes, hipMemcpyDeviceToHost);

        for (int i = 0; i < nBodies; i++) { // integrate position
            p[i].x += p[i].vx * dt;
            p[i].y += p[i].vy * dt;
      
        }

        const double tElapsed = GetTimer() / 1000.0;
        if (iter > 1) { // First iter is warm up
            totalTime += tElapsed;
        }
#ifndef SHMOO
        printf("Iteration %d: %.3f seconds\n", iter, tElapsed);
#endif
    }
    double avgTime = totalTime / (double)(nIters - 1);

#ifdef SHMOO
    printf("%d, %0.3f\n", nBodies, 1e-9 * nBodies * nBodies / avgTime);
#else
    //printf("Average rate for iterations 2 through %d: %.3f +- %.3f steps per second.\n",nIters, rate);
    printf("%d Bodies: average %0.3f Billion Interactions / second\n", nBodies, 1e-9 * nBodies * nBodies / avgTime);
#endif
    free(buf);
    hipFree(d_buf);
}

