#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""
#include "Particle.h"

#include <math.h>



__device__ float vecMag(glm::vec3 a) {
    return sqrt(a[0] * a[0] + a[1] * a[1] + a[2] * a[2]);
}

__device__ glm::vec3 unitVec(glm::vec3 a) {
    return a / vecMag(a);
}



__global__ void solve(glm::vec3* cPos, glm::vec3* pPos, glm::vec3* nPos, float* mass, float dt, unsigned int _n, float softening = 0.01) {
    float G = 1;
    int index = threadIdx.x * blockIdx.x;
    for (int i = 0; i<int(_n); i++) {
        glm::vec3 distance = cPos[i] - cPos[index];

    }


}










void particleSystem::updateSystem(float dt) {
    
    //Set up GPU
    glm::vec3* gpuPPos;
    glm::vec3* gpuNPos;
    glm::vec3* gpuCPos;
    float* mass;
   
    hipSetDevice(0);
    
    //Allocate memory
    hipMalloc((void**)gpuCPos, n * 3 * sizeof(float));
    hipMalloc((void**)gpuPPos, n * 3 * sizeof(float));
    hipMalloc((void**)gpuNPos, n * 3 * sizeof(float));
    
    //Set Values
    hipMemcpy(gpuCPos, &currentPosition[0], n * sizeof(glm::vec3), hipMemcpyHostToDevice);
    hipMemcpy(gpuPPos, &previousPosition[0], n * sizeof(glm::vec3), hipMemcpyHostToDevice);
    
    solve <<<1, n>>>(gpuCPos, gpuPPos, gpuNPos, mass, dt, n);
    
    hipDeviceSynchronize(); //Wait for processes to finish
    
    std::copy(&previousPosition.begin(), &previousPosition.end(), &currentPosition.begin()); //current position is now previousPosition
    hipMemcpy(&currentPosition, gpuNPos, n * sizeof(glm::vec3), hipMemcpyDeviceToHost);

    return;
}