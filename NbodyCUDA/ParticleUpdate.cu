#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""
#include "Particle.h"
#include <stdio.h>
#include <math.h>
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

int main1()
{
    const int arraySize = 5;
    const int a[arraySize] = { 1, 2, 3, 4, 5 };
    const int b[arraySize] = { 10, 20, 30, 40, 50 };
    int c[arraySize] = { 0 };

    // Add vectors in parallel.
    hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

    printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
        c[0], c[1], c[2], c[3], c[4]);

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    addKernel<<<1, size>>>(dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}



__device__ float vecMag(glm::vec3 a) {
    return sqrt(a[0] * a[0] + a[1] * a[1] + a[2] * a[2]);
}

__device__ glm::vec3 unitVec(glm::vec3 a) {
    return a / vecMag(a);
}



__global__ void solve(glm::vec3* cPos, glm::vec3* pPos, glm::vec3* nPos, float* mass, float dt, unsigned int _n, float softening = 0.01) {
    float G = 1;
    int id = threadIdx.x * blockIdx.x;
    int n = int(_n);
    glm::vec3  tAccel = glm::vec3(0.0, 0.0, 0.0);
    for (int i = 0; i < n; i++) {
        if (i != id) {
            glm::vec3 distance = cPos[i] - cPos[id];
            float mag = vecMag(distance);
            glm::vec3 accel = (G * mass[i]/(mag * mag)) * unitVec(distance);
            tAccel += accel;
        }
    }

    nPos[id] = cPos[id] + cPos[id] - pPos[id] + (tAccel * dt * dt);


}











void particleSystem::updateSystem(float dt) {
    //Set up GPU
    glm::vec3* gpuCPos;
    glm::vec3* gpuPPos;
    glm::vec3* gpuNPos;
    float* mass;
   
    hipSetDevice(0);

    //Allocate memory
    hipMalloc((void**)gpuCPos, n * sizeof(glm::vec3));
    hipMalloc((void**)gpuPPos, n * sizeof(glm::vec3));
    hipMalloc((void**)gpuNPos, n * sizeof(glm::vec3));

    //Set Values
    hipMemcpy(gpuCPos, &currentPosition[0].x, n * sizeof(glm::vec3), hipMemcpyHostToDevice);
    hipMemcpy(gpuPPos, &previousPosition[0].x, n * sizeof(glm::vec3), hipMemcpyHostToDevice);

    solve <<<1, n>>>(gpuCPos, gpuPPos, gpuNPos, mass, dt, n);
    hipDeviceSynchronize(); //Wait for processes to finish
    std::copy(currentPosition[0], currentPosition[n], previousPosition); //current position is now previousPosition
    hipMemcpy(&currentPosition, gpuNPos, n * sizeof(glm::vec3), hipMemcpyDeviceToHost);

    //
}